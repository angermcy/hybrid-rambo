#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <stdio.h>
#include <iomanip>

#include "BloomFilterGPU.cuh"
#include "constants.h"
#include "bitArray.h"
#include "deviceFctsGPU.cuh"
#include "RamboGPU.cuh"
#include "utils.h"
#include "bitComputationGPU.cuh"

/*
 *  global variables
 */

// GPU pointers
int *gpuMRs;
uint *gpuMRSizes;
uint *gpuMROffsets;
char *gpuBait;
char *gpuBar;
uint *gpuTP;
uint *gpuTB;

// CUDA streams
hipStream_t strm_SetBit;
hipStream_t strm_SetBarFV;
hipStream_t strm_SetBarAV;
hipStream_t strm_CopyBack;
hipStream_t strm_clnBar;
hipStream_t strm_clnBait;

hipStream_t strm_acTP;
hipStream_t strm_cpTP;
hipStream_t strm_acTB;
hipStream_t strm_cpTB;

// for internal tests only
/*
int  *cpuMRs;
uint *cpuMRSizes;
uint *cpuMROffsets;
char *cpuBait;
char *cpuBar;
*/

/*
 *
 */
void GPUCopyMRtoGPU(int n, int rCols, int rRows, int barSize, std::vector<int>* metaRambo, uint *mrSize, uint *mrSizesOffsSize)
{

    //hipStream_t strm_cpRA;
    hipStream_t strm_cpMRs;
    hipStream_t strm_cpMRSizes;
    hipStream_t strm_cpMROffsets;

    std::vector<uint> mRSizes;
    std::vector<uint> mROffsets;
    std::vector<int> mRs;

    // reset all vectors
    mRs.clear();
    mRSizes.clear();
    mROffsets.clear();

    uint totSize=0;
    int i;
    //int rangeRA = n/8+1;

    //hipStreamCreate(&strm_cpRA);
    //CudaErr("hipMalloc/GPUCopyMRtoGPU/gpuRA", hipMalloc((void **)&gpuRA, rCols*rRows*rangeRA*sizeof(char)));
    //std::cout << "gpuRA size: " << (rCols*rRows*rangeRA*sizeof(char)) << " " << rangeRA << std::endl;

    // copies all metaRambo data into vectors - one big vector
    for (int c = 0; c < rCols; c++)
    {
        for (int r = 0; r < rRows; r++)
        {
            i=r+rRows*c;
            std::copy(metaRambo[i].begin(), metaRambo[i].end(), std::back_inserter(mRs)); // whole metaRambo at idx i copied into mRs
            mRSizes.push_back(metaRambo[i].size()); // size of metaRambo at idx i
            mROffsets.push_back(totSize); //position in bitArray
            //std::cout << "MR: " << c << " r: " << r << " pos: " << (c*rRows)+r << " offs: " << totSize << " size: " << metaRambo[i].size() << std::endl;
            totSize += metaRambo[i].size(); // offset counter

            //std::cout << "gpuRA cp c: " << c << " r: " << r << " pos: " << (c*r*rangeRA*sizeof(char)) << " max: " << (rCols*rRows*rangeRA*sizeof(char)) << std::endl;
            //CudaErr("hipMemcpyAsync/GPUCopyMRtoGPU/gpuRA", hipMemcpyAsync(gpuRA+(c*r*rangeRA*sizeof(char)), aBF[c*r]->m_bits->A, rangeRA*sizeof(char), hipMemcpyHostToDevice, strm_cpRA));

        }
    }

    // calculate sizes of vectors
    *mrSize = totSize * sizeof(int);
    *mrSizesOffsSize = mRSizes.size() * sizeof(uint);

    /*
     *  copies big data values onto GPU
     */
    // create CUDA streams
    hipStreamCreate(&strm_cpTP);
    hipStreamCreate(&strm_cpMRs);
    hipStreamCreate(&strm_cpMRSizes);
    hipStreamCreate(&strm_cpMROffsets);

    hipStreamCreate(&strm_SetBit);
    hipStreamCreate(&strm_SetBarFV);
    hipStreamCreate(&strm_SetBarAV);
    hipStreamCreate(&strm_CopyBack);

    hipStreamCreate(&strm_clnBar);
    hipStreamCreate(&strm_clnBait);

    // CUDA malloc
    CudaErr("hipMalloc/GPUCopyMRtoGPU/gpuTB", hipMalloc((void **)&gpuTB, 2*rCols*sizeof(uint)));  // rCols long but rCols consists of 2 int per entry
    CudaErr("hipMalloc/GPUCopyMRtoGPU/gpuTP", hipMalloc((void **)&gpuTP, rCols*rRows*sizeof(uint)));

    CudaErr("hipMalloc/GPUCopyMRtoGPU/gpuMRs", hipMalloc((void **)&gpuMRs, totSize * sizeof(int)));
    CudaErr("hipMalloc/GPUCopyMRtoGPU/gpuMRSizes", hipMalloc((void **)&gpuMRSizes, mRSizes.size() * sizeof(uint)));
    CudaErr("hipMalloc/GPUCopyMRtoGPU/gpuMROffsets", hipMalloc((void **)&gpuMROffsets, mROffsets.size() * sizeof(uint)));

    CudaErr("hipMalloc/GPUCopyMRtoGPU/gpuBait", hipMalloc((void **)&gpuBait, barSize*rCols*rRows*sizeof(char))); // bait w. max size "RxBxKi"
    CudaErr("hipMalloc/GPUCopyMRtoGPU/gpuBar", hipMalloc((void **)&gpuBar, barSize*sizeof(char)));

    // CUDA memcpyasync
    CudaErr("hipMemcpyAsync/GPUCopyMRtoGPU/gpuMRs", hipMemcpyAsync(gpuMRs, mRs.data(), totSize*sizeof(int), hipMemcpyHostToDevice, strm_cpMRs));
    CudaErr("hipMemcpyAsync/GPUCopyMRtoGPU/gpuMRSizes", hipMemcpyAsync(gpuMRSizes, mRSizes.data(), mRSizes.size()*sizeof(uint), hipMemcpyHostToDevice, strm_cpMRSizes));
    CudaErr("hipMemcpyAsync/GPUCopyMRtoGPU/gpuMROffsest", hipMemcpyAsync(gpuMROffsets, mROffsets.data(), mROffsets.size()*sizeof(uint), hipMemcpyHostToDevice, strm_cpMROffsets));

    // synchronize
    CudaErr("hipDeviceSynchronize/GPUCopyMRtoGPU", hipDeviceSynchronize());

    // destroy CUDA streams
    //CudaErr("hipStreamDestroy/GPUCopyMRtoGPU/strm_cpRA", hipStreamDestroy(strm_cpRA));
    CudaErr("hipStreamDestroy/GPUCopyMRtoGPU/strm_cpMRs", hipStreamDestroy(strm_cpMRs));
    CudaErr("hipStreamDestroy/GPUCopyMRtoGPU/strm_cpMRSizes", hipStreamDestroy(strm_cpMRSizes));
    CudaErr("hipStreamDestroy/GPUCopyMRtoGPU/strm_cpMROffsets", hipStreamDestroy(strm_cpMROffsets));

    /*
        // FOR EVALUATION IF COPY TO CPU IS CORRECT
        hipStream_t strm_test1;
        hipStreamCreate(&strm_test1);

        cpuMRs=(int*)malloc(totSize*sizeof(int));
        CudaErr("tst", hipMemcpyAsync(cpuMRs, gpuMRs, totSize*sizeof(int), hipMemcpyDeviceToHost, strm_test1));
        CudaErr("tst", hipDeviceSynchronize());
        CudaErr("tst", hipStreamDestroy(strm_test1));

        int z=0;
        for (int i=0; i < totSize; i++ ) {

            if (mRs[i] != cpuMRs[i]) {
                z++;
                std::cout << "pos " << i << " " << mRs[i] << " <> " << cpuMRs[i] << std::endl;
            }
        }
        std::cout << "MRs found " << z << " unidentical ints of " << totSize << std::endl;

        hipStream_t strm_test2;
        hipStreamCreate(&strm_test2);
        hipStream_t strm_test3;
        hipStreamCreate(&strm_test3);

        cpuMRSizes=(uint*)malloc(mRSizes.size()*sizeof(uint));
        CudaErr("tst", hipMemcpyAsync(cpuMRSizes, gpuMRSizes, mRSizes.size()*sizeof(uint), hipMemcpyDeviceToHost, strm_test2));

        cpuMROffsets=(uint*)malloc(mROffsets.size()*sizeof(uint));
        CudaErr("tst", hipMemcpyAsync(cpuMROffsets, gpuMROffsets, mROffsets.size()*sizeof(uint), hipMemcpyDeviceToHost, strm_test3));

        CudaErr("tst", hipDeviceSynchronize());
        CudaErr("tst", hipStreamDestroy(strm_test2));
        CudaErr("tst", hipStreamDestroy(strm_test3));

        for (int i=0; i < mRSizes.size(); i++ ) {
            std::cout << "pos " << i << " (";
            std::cout << mRSizes[i] << "=";
            std::cout << cpuMRSizes[i] << ") offs (";
            std::cout << mROffsets[i] << "=";
            std::cout << cpuMROffsets[i] << ")" << std::endl;
        }

        cpuBait = (char *)malloc(barSize*rCols*rRows* sizeof(char));
        cpuBar = (char *)malloc(barSize * sizeof(char));
        */

}

/*
 *  initalises the GPU pointers with '\0'
 */
void GPUEmptyBarAndBait(int barSize, int baitSize)
{

    CudaErr("hipMemsetAsync/GPUCopyMRtoGPU/gpuBar", hipMemsetAsync(gpuBar, '\0', barSize*sizeof(char), strm_clnBar));
    CudaErr("hipMemsetAsync/GPUCopyMRtoGPU/gpuBait", hipMemsetAsync(gpuBait, '\0', baitSize*sizeof(char), strm_clnBait));
    CudaErr("hipDeviceSynchronize/GPUCopyMRtoGPU", hipDeviceSynchronize());

    // TEST
    /*
    memset(cpuBar, '\0', barSize*sizeof(char) );
    memset(cpuBait, '\0', baitSize*sizeof(char) );
    */
}

/*
 *  copies all test data onto the GPU
 */
void GPUCopyTestPositionsToGPU(std::vector<uint>& tp, std::vector<uint>& tb)
{
    CudaErr("hipMemcpyAsync/GPUCopyTestPositionsToGPU/gpuTP", hipMemcpyAsync(gpuTP,
            tp.data(),
            tp.size()*sizeof(uint),
            hipMemcpyHostToDevice, strm_cpTP));

    CudaErr("hipMemcpyAsync/GPUCopyTestPositionsToGPU/gpuTB", hipMemcpyAsync(gpuTB,
            tb.data(),
            tb.size()*sizeof(uint),
            hipMemcpyHostToDevice, strm_cpTB));
    CudaErr("hipDeviceSynchronize/GPUCopyTestPositionsToGPU/gpuTP", hipDeviceSynchronize());
    /*
        uint *cpuPos;
        cpuPos = (uint *)malloc(tp.size()*sizeof(uint));
        hipStream_t strm_test2;
        hipStreamCreate(&strm_test2);

        CudaErr("hipMemcpyAsync/GPUCopyTestPositionsToGPU/gpuTP", hipMemcpyAsync(cpuPos,
                                                                                   gpuTP,
                                                                                   tp.size()*sizeof(uint),
                                                                                   hipMemcpyDeviceToHost, strm_test2));
        CudaErr("hipDeviceSynchronize/GPUCopyTestPositionsToGPU/gpuTP", hipDeviceSynchronize());

        CudaErr("tst", hipStreamDestroy(strm_test2));

        for (uint i=0; i < tp.size()/2; i++ ) {

            std::cout << "size (" << tp[i*2] << "=" << cpuPos[i*2] << ") (" << tp[i*2+1] << "=" << cpuPos[i*2+1] << ")" << std::endl;
        }
    */
}

/*
 *  destroys all used streams and pointers
 */
void GPUCleanupMR(void)
{
    CudaErr("hipStreamDestroy/GPUCleanupMR/streamMRs", hipStreamDestroy(strm_SetBit));
    CudaErr("hipStreamDestroy/GPUCleanupMR/strm_SetBarFV", hipStreamDestroy(strm_SetBarFV));
    CudaErr("hipStreamDestroy/GPUCleanupMR/strm_SetBarFV", hipStreamDestroy(strm_SetBarAV));
    CudaErr("hipStreamDestroy/GPUCleanupMR/strm_CopyBack", hipStreamDestroy(strm_CopyBack));

    CudaErr("hipFree/GPUCleanupMR/gpuMRs", hipFree(gpuMRs));
    CudaErr("hipFree/GPUCleanupMR/gpuMRSizes", hipFree(gpuMRSizes));
    CudaErr("hipFree/GPUCleanupMR/gpuMROffsets", hipFree(gpuMROffsets));

    CudaErr("hipStreamDestroy/GPUCopyMRtoGPU/strm_clnBar", hipStreamDestroy(strm_clnBar));
    CudaErr("hipStreamDestroy/GPUCopyMRtoGPU/strm_clnBait", hipStreamDestroy(strm_clnBait));
}

/*
 *  calculates the column-wise union of char-arrays (Bloom Filter for Union) to compromise the data
 *  SetBitAllGpu: bitwise OR
 */
void GPURunSetBit(uint numBaits, uint barSize)
{
    dim3 gridSize(1);
    dim3 blockSize(numBaits); //no of threads

    SetBitAllGpu<<<gridSize, blockSize>>>(barSize, gpuTB, gpuTP, gpuBait, gpuMRs, gpuMRSizes, gpuMROffsets);
    CudaErr("hipDeviceSynchronize/GPURunSetBit2", hipDeviceSynchronize());
}

/*
 *  sets the first value for bitarray result (bar) with first char-array of bitarray interim results (bait)
 *  calculates intersection of bar and left over char-arrays from bait
 *  result: char-array (1:= documents at this position in metarambo potentially contains query k-mer)
 */
void GPUCombineBaitToBar(uint barSize, uint baitCols)
{
    SetBarFirstValueGpu<<<1, 1, 0, strm_SetBarFV>>>(gpuBar, gpuBait, barSize);
    CudaErr("hipDeviceSynchronize/GPUGetResult 1", hipDeviceSynchronize());

    dim3 gridSize(1);
    dim3 blockSize(baitCols); //no of threads

    AndOpAllGpu<<<gridSize, blockSize>>>(gpuBar, gpuBait, baitCols, barSize);
    CudaErr("hipDeviceSynchronize/GPUGetResult 2", hipDeviceSynchronize());
}


/*
 *  saves GPU calculated result in bitArray and returns bitArray
 */
bitArray GPUGetResult(uint K, uint barSize)
{
    bitArray resBar(K);

    /*  TEST FCT TO RETURN BITARRAY
        char *x;
        char *y;

        x=(char*)malloc(baitCols*barSize*sizeof(char));
        y=(char*)malloc(barSize*sizeof(char));

        CudaErr("16 - hipMemcpyAsync/(resBar.A", hipMemcpyAsync(x, gpuBait, baitCols*barSize*sizeof(char), hipMemcpyDeviceToHost, strm_CopyBack));
        CudaErr("hipDeviceSynchronize/GPUGetResult 3", hipDeviceSynchronize());

        int z=0;
        for (int i=0; i < baitCols*barSize*sizeof(char); i++ ) {
            if (z >= barSize) {
                z=0;
                std::cout << std::endl;
            }
            z++;
            std::cout << atoi( &x[i] ) << ", ";
        }

        // reduce bait to bitarray
        std::cout << "SetBarFirst: " << barSize << std::endl;
    */


    CudaErr("hipMemcpyAsync/(resBar.A", hipMemcpyAsync(resBar.A, gpuBar, barSize*sizeof(char), hipMemcpyDeviceToHost, strm_CopyBack));
    CudaErr("hipDeviceSynchronize/GPUGetResult 3", hipDeviceSynchronize());

    return resBar;
}


/*
 *  prints out last CUDA errors with specific text
 */
void CudaErrorPrint(std::string txt)
{
    hipError_t hipError_t;

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        std::cout << txt << ", last cuda error: " << hipGetErrorString(hipError_t) << std::endl;
    }
    /*else
    {
        std::cout << txt << ", no error." << std::endl;
    }*/

}

/*
 *  prints out function specific CUDA error with specific text
 */
void CudaErr(std::string msg, hipError_t hipError_t)
{
    if (hipError_t != hipSuccess)
    {
        std::cout << "CUDA ERROR at " << msg << ": " << hipGetErrorString(hipError_t) << std::endl;
    }
}

