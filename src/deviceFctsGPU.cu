#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "bitArray.h"
#include "deviceFctsGPU.cuh"

/*
 *  union of all true document char-arrays per column into interium result bait
 *  params: barSize, testBait, testPositions, bait, mR, mRSizes, mROffsets (from GPUCopyMRtoGPU in bitComputationGPU)

         example:
         testBait size is 2*r (maxColumns from MetaRambo)
         0 -> start     - thread 1 r=0
         3 -> offset    - thread 1 r=0
         3 -> start     - thread 2 r=1
         4 -> offset    - thread 2 r=1
         7 -> start     - thread 3 r=2
         3 -> offset    - thread 3 r=2
         ...

         testPositions
         5 - candidate 1 for thread 1     (start idx = 0)
         7
         9
         17 - candidate 1 for thread 2    (start idx = 3)
         18
         22
         24
         36 - candidate 1 for thread 3    (start idx = 7)
         39
         40
         ...
 */
__global__ void SetBitAllGpu(uint barSize, uint *tb, uint *tp, char *bait, int *mr, uint *mrSize, uint *mrOffs)
{
    int i = threadIdx.x; // this replaces the for loop  ->  no of threadIdx = numBaits

    //for (uint i = 0; i <= numBaits; i++)
    for (uint t=tb[i*2]; t<tb[i*2]+tb[i*2+1]; t++)
    {
        // for test purposes
        //printf("bait pos loop t=%d posinTP0: %d PosInTP1: %d  bp0: %d bp1: %d Offs=%d Sizefs=%d \n", t,
        //       tp[t*2],
        //       tp[t*2],
        //       mrOffs[ tp[ t*2 ] ],
        //       mrSize[ tp[ t*2 ] ]
        //       );

        for (uint s = 0; s < mrSize[ tp[ t ] ]; s++)
        {
            bait[(i*barSize) + (mr[mrOffs[ tp[ t ] ] + s] / 8)] |= (1 << (mr[mrOffs[ tp[ t ] ] + s] % 8));
        }
    }
    //}

    return;
}


/*
 *  initialise bitarray result (bar) with first char-Array of bitarray interim result (bait)
 */
__global__ void SetBarFirstValueGpu(char *bar, char *bait, int bitarrayLength)
{
    for (int len = 0; len < bitarrayLength; len++)
    {
        bar[len] = bait[len]; //+ 1];
    }
}


/*
 *  intersection/AND-join of all interium result char-arrays (per column) into result bar
 */
__global__ void AndOpAllGpu(char *bar, char *bait, int baitCols, int bitarrayLength)
{
    int baitNumber = threadIdx.x; // replaces the loop
    //for (int baitNumber = index; baitNumber < baitCols; baitNumber+=stride)
    //{
    for (int len = 0; len < bitarrayLength; len++)
    {
        bar[len] &= bait[bitarrayLength * baitNumber + len]; //+ 1];
    }
    //}
}
